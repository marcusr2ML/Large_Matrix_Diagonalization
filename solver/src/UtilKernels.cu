#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <algorithm>
#include "CudaHelper.h"
#include <cstdio>

namespace GPU {

template <typename Scalar>
__global__ void permuteKernel(const Scalar* __restrict__ v, const int* __restrict__ perm, Scalar* __restrict__ dst, int size)
{
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    dst[perm[i]] = v[i];
  }
}

template <typename Scalar>
__global__ void addInPlaceKernel(Scalar * __restrict__ lhs, Scalar * __restrict__ rhs)
{
  *lhs += *rhs;
}

template <>
__global__ void addInPlaceKernel<hipComplex>(hipComplex *__restrict__ a, hipComplex *__restrict__ b)
{
  a->x += b->x;
  a->y += b->y;
}

template <>
__global__ void addInPlaceKernel<hipDoubleComplex>(hipDoubleComplex *__restrict__ a, hipDoubleComplex *__restrict__ b)
{
  a->x += b->x;
  a->y += b->y;
}

template <typename Scalar, typename RealType>
__global__ void divideByRealKernel(const Scalar * __restrict__ v, const RealType * __restrict__ s, Scalar * __restrict__ res, int size) {
  RealType scalar = *s;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    res[i] = v[i] / scalar;
  }
}

template <>
__global__ void divideByRealKernel<hipComplex, float>(const hipComplex * __restrict__ v, const float * __restrict__ s, hipComplex * __restrict__ res, int size) {
  float scalar = *s;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    res[i].x = v[i].x / scalar;
    res[i].y = v[i].y / scalar;
  }
}

template <>
__global__ void divideByRealKernel<hipDoubleComplex, double>(const hipDoubleComplex * __restrict__ v, const double * __restrict__ s, hipDoubleComplex * __restrict__ res, int size) {
  double scalar = *s;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    res[i].x = v[i].x / scalar;
    res[i].y = v[i].y / scalar;
  }
}


template <typename Scalar>
__global__ void eigshNormalizeKernel(Scalar * __restrict__ col, Scalar * __restrict__ v, int n,
                                     const Scalar * __restrict__ u ,const Scalar * __restrict__ beta)
{
  Scalar b = *beta;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    Scalar res = u[i] / b;
    col[i] = res;
    v[i] = res;
  }
}

template <>
__global__ void eigshNormalizeKernel<hipComplex>(hipComplex * __restrict__ col, hipComplex * __restrict__ v, int n,
                                                const hipComplex * __restrict__ u ,const hipComplex * __restrict__ beta)
{
  float b = beta->x;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    hipComplex res = make_hipComplex(u[i].x / b, u[i].y / b);
    col[i] = res;
    v[i] = res;
  }
}

template <>
__global__ void eigshNormalizeKernel<hipDoubleComplex>(hipDoubleComplex * __restrict__ col, hipDoubleComplex * __restrict__ v, int n,
                                                      const hipDoubleComplex * __restrict__ u ,const hipDoubleComplex * __restrict__ beta)
{
  double b = beta->x;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    hipDoubleComplex res = make_hipDoubleComplex(u[i].x / b, u[i].y / b);
    col[i] = res;
    v[i] = res;
  }
}

void _permute(float *v, const int *perm, void *buffer, int size)
{
  using Scalar = float;
  permuteKernel<Scalar><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, perm, (Scalar *) buffer, size);
  CHECK_CUDA( hipMemcpy(v, buffer, size * sizeof(Scalar), hipMemcpyDeviceToDevice) );
}

void _permute(double *v, const int *perm, void *buffer, int size)
{
  using Scalar = double;
  permuteKernel<Scalar><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, perm, (Scalar *) buffer, size);
  CHECK_CUDA( hipMemcpy(v, buffer, size * sizeof(Scalar), hipMemcpyDeviceToDevice) );
}

void _permute(hipComplex *v, const int *perm, void *buffer, int size)
{
  using Scalar = hipComplex;
  permuteKernel<Scalar><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, perm, (Scalar *) buffer, size);
  CHECK_CUDA( hipMemcpy(v, buffer, size * sizeof(Scalar), hipMemcpyDeviceToDevice) );
}

void _permute(hipDoubleComplex *v, const int *perm, void *buffer, int size)
{
  using Scalar = hipDoubleComplex;
  permuteKernel<Scalar><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, perm, (Scalar *) buffer, size);
  CHECK_CUDA( hipMemcpy(v, buffer, size * sizeof(Scalar), hipMemcpyDeviceToDevice) );
}

void addInPlace(float * __restrict__ lhs, float * __restrict__ rhs) {
  addInPlaceKernel<float><<<1, 1>>>(lhs, rhs);
}

void addInPlace(double * __restrict__ lhs, double * __restrict__ rhs) {
  addInPlaceKernel<double><<<1, 1>>>(lhs, rhs);
}

void addInPlace(hipComplex * __restrict__ lhs, hipComplex * __restrict__ rhs) {
  addInPlaceKernel<hipComplex><<<1, 1>>>(lhs, rhs);
}

void addInPlace(hipDoubleComplex * __restrict__ lhs, hipDoubleComplex * __restrict__ rhs) {
  addInPlaceKernel<hipDoubleComplex><<<1, 1>>>(lhs, rhs);
}

void eigshNormalize(float * __restrict__ col, float * __restrict__ v, int n,
                    const float * __restrict__ u ,const float * __restrict__ beta)
{
  eigshNormalizeKernel<float><<<std::min((int) std::ceil(n / 512.0f), 80), 512>>>(col, v, n, u, beta);
}

void eigshNormalize(double * __restrict__ col, double * __restrict__ v, int n,
                    const double * __restrict__ u ,const double * __restrict__ beta)
{
  eigshNormalizeKernel<double><<<std::min((int) std::ceil(n / 512.0f), 80), 512>>>(col, v, n, u, beta);
}

void eigshNormalize(hipComplex * __restrict__ col, hipComplex * __restrict__ v, int n,
                    const hipComplex * __restrict__ u ,const hipComplex * __restrict__ beta)
{
  eigshNormalizeKernel<hipComplex><<<std::min((int) std::ceil(n / 512.0f), 80), 512>>>(col, v, n, u, beta);
}

void eigshNormalize(hipDoubleComplex * __restrict__ col, hipDoubleComplex * __restrict__ v, int n,
                    const hipDoubleComplex * __restrict__ u ,const hipDoubleComplex * __restrict__ beta)
{
  eigshNormalizeKernel<hipDoubleComplex><<<std::min((int) std::ceil(n / 512.0f), 80), 512>>>(col, v, n, u, beta);
}

void _divideByReal(const float * __restrict__ v, const float * __restrict__ s, float * __restrict__ res, int size) {
  divideByRealKernel<float, float><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, s, res, size);
}

void _divideByReal(const double * __restrict__ v, const double * __restrict__ s, double * __restrict__ res, int size) {
  divideByRealKernel<double, double><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, s, res, size);
}

void _divideByReal(const hipComplex * __restrict__ v, const float * __restrict__ s, hipComplex * __restrict__ res, int size) {
  divideByRealKernel<hipComplex, float><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, s, res, size);
}

void _divideByReal(const hipDoubleComplex * __restrict__ v, const double * __restrict__ s, hipDoubleComplex * __restrict__ res, int size) {
  divideByRealKernel<hipDoubleComplex, double><<<std::min((int) std::ceil(size / 512.0f), 80), 512>>>(v, s, res, size);
}

} // Namespace GPU
